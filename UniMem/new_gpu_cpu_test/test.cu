#include <hip/hip_runtime.h>
#include <iostream>

// Define the REAL type as float
using REAL = float;

// Kernel to perform repeated accesses to the unified memory
__global__ void UnifiedMemoryAccessKernel(REAL* x, REAL* y, int n, REAL a, int stride) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < (n / stride)) {
        for (int i = 0; i < 1000; ++i) {
            y[idx] = a * x[idx * stride];
        }
    }
}

void MemoryAccessTest(REAL* x, REAL* y, int n, REAL a, int stride) {
    REAL *d_x, *d_y;
    hipMallocManaged(&d_x, n * sizeof(REAL));
    hipMallocManaged(&d_y, (n / stride) * sizeof(REAL));

    // Initialize the input array on the host
    for (int i = 0; i < n; ++i) {
        x[i] = static_cast<REAL>(i);
    }

    // Copy data from host to device (not strictly necessary with managed memory)
    hipMemcpy(d_x, x, n * sizeof(REAL), hipMemcpyHostToDevice);

    // Define the number of threads and blocks
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;

    // Call the kernel to access the list on the GPU
    for (int i = 0; i < 100; ++i) { // Accessing it many times
        UnifiedMemoryAccessKernel<<<blocksPerGrid, threadsPerBlock>>>(d_x, d_y, n, a, stride);
        hipDeviceSynchronize();
    }

    // Optionally copy the result back to the host
    hipMemcpy(y, d_y, (n / stride) * sizeof(REAL), hipMemcpyDeviceToHost);

    // Free the unified memory
    hipFree(d_x);
    hipFree(d_y);
}

int main() {
    const int n = 134217728; // Size of the list
    const REAL a = 2.0f; // Multiplier
    const int stride = 1; // Stride value
    REAL *x = new REAL[n];
    REAL *y = new REAL[n / stride];
    int num_runs = 3;
    //warm up
    MemoryAccessTest(x, y, n, a, stride);

    // Run the test
    for (int i=0; i<num_runs; i++) MemoryAccessTest(x, y, n, a, stride);
  
    // Check the results
    for (int i = 0; i < 10; ++i) {
        std::cout << y[i] << " ";
    }
    std::cout << std::endl;

    delete[] x;
    delete[] y;

    return 0;
}
